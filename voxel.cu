#include "hip/hip_runtime.h"
//
//  voxel.cu
//  optixVolumetric
//
//  Created by Tim Tavlintsev (TVL)
//
//

#include <optix_world.h>

using namespace optix;

rtBuffer<float4> voxel_buffer;
rtBuffer<float4> color_buffer;

rtDeclareVariable(float4,  sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// object color assigned at intersection time
rtDeclareVariable(float4, obj_color, attribute obj_color, );
rtDeclareVariable(float, cutoff_from, , );
rtDeclareVariable(float, cutoff_to, , );

//
// Box
//
static __device__ void make_box(const float4 & input, float3 & boxmin, float3  & boxmax) {
  float halfWidth = input.w/2;
  boxmin.x = input.x-halfWidth; boxmax.x = input.x+halfWidth;
  boxmin.y = input.y-halfWidth; boxmax.y = input.y+halfWidth;
  boxmin.z = input.z-halfWidth; boxmax.z = input.z+halfWidth;
}

static __device__ float3 boxnormal(const float3 & boxmin, const float3 & boxmax, const float & t)
{
  
  float3 t0 = (boxmin - ray.origin)/ray.direction;
  float3 t1 = (boxmax - ray.origin)/ray.direction;
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

RT_PROGRAM void box_intersect(int primIdx)
{
  // filter 0 values and colors cutted by cutoff_from/cutoff_to
  if (color_buffer[primIdx].w == 0) return;
  else if (color_buffer[primIdx].w < cutoff_from || color_buffer[primIdx].w > cutoff_to) return;

	float3 boxmin, boxmax;
	make_box(voxel_buffer[primIdx], boxmin, boxmax);

	float3 t0 = (boxmin - ray.origin)/ray.direction;
  float3 t1 = (boxmax - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
       shading_normal = geometric_normal = boxnormal(boxmin, boxmax, tmin );
       obj_color = color_buffer[primIdx];
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
        shading_normal = geometric_normal = boxnormal(boxmin, boxmax, tmax );
        obj_color = color_buffer[primIdx];
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void box_bounds (int primIdx, float result[6])
{
	float3 boxmin, boxmax;
	make_box(voxel_buffer[primIdx], boxmin, boxmax);
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}

//
// Sphere 
//

RT_PROGRAM void sphere_intersect(int primIdx)
{
  // filter 0 values and colors cutted by cutoff_from/cutoff_to
  if (color_buffer[primIdx].w == 0) return;
  else if (color_buffer[primIdx].w < cutoff_from || color_buffer[primIdx].w > cutoff_to) return;

  float3 center = make_float3(voxel_buffer[primIdx]);
  float radius = voxel_buffer[primIdx].w/2;

  float3 V = center - ray.origin;
  float b = dot(V, ray.direction);
  float disc = b*b + radius*radius - dot(V, V);
  if (disc > 0.0f) {
    disc = sqrtf(disc);

#define FASTONESIDEDSPHERES 1
#if defined(FASTONESIDEDSPHERES)
    // only calculate the nearest intersection, for speed
    float t1 = b - disc;
    if (rtPotentialIntersection(t1)) {
      shading_normal = geometric_normal = (t1*ray.direction - V) / radius;
      obj_color = color_buffer[primIdx]; // uniform color for the entire object
      rtReportIntersection(0);
    }
#else
    float t2 = b + disc;
    if (rtPotentialIntersection(t2)) {
      shading_normal = geometric_normal = (t2*ray.direction - V) / radius;
      // float3 offset = shading_normal * scene_epsilon;
      obj_color = color_buffer[primIdx]; // uniform color for the entire object
      rtReportIntersection(0);
    }

    float t1 = b - disc;
    if (rtPotentialIntersection(t1)) {
      shading_normal = geometric_normal = (t1*ray.direction - V) / radius;
      obj_color = color_buffer[primIdx]; // uniform color for the entire object
      rtReportIntersection(0);
    }
#endif
  }
}



RT_PROGRAM void sphere_bounds (int primIdx, float result[6])
{
  const float3 cen = make_float3( voxel_buffer[primIdx] );
  const float3 rad = make_float3( voxel_buffer[primIdx].w/2 );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}