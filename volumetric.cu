#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software, related documentation and any
 * modifications thereto.  Any use, reproduction, disclosure or distribution of
 * this software and related documentation without an express license agreement
 * from NVIDIA Corporation is strictly prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED
 * *AS IS* AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS
 * OR IMPLIED, INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF
 * MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.  IN NO EVENT SHALL
 * NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY SPECIAL, INCIDENTAL, INDIRECT, OR
 * CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT LIMITATION, DAMAGES FOR
 * LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF BUSINESS
 * INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGES
 */

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};


rtBuffer<BasicLight>                 lights;
rtDeclareVariable(float3,            ambient_light_color, , );
rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );
rtDeclareVariable(float,             scene_epsilon, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 
rtDeclareVariable(float4, obj_color, attribute obj_color, );

rtDeclareVariable(float3, shadow_attenuation, , );
rtDeclareVariable(int, show_shadows, , );

rtDeclareVariable(optix::Ray,          ray,        rtCurrentRay, );
rtDeclareVariable(float,               t_hit,      rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd,        rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{

// #define TRANSPARENT 1
#ifndef TRANSPARENT
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = optix::make_float3(0);
  rtTerminateRay();

#else
  // Attenuates shadow rays for shadowing transparent objects
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
#endif

}


RT_PROGRAM void closest_hit_radiance()
{
  float3 world_shading_normal = optix::normalize( 
      rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = optix::normalize( 
      rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 normal = optix::faceforward(
      world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hit_point = ray.origin + t_hit * ray.direction;
  
  float3 Kd = make_float3( obj_color ); 

  // ambient contribution
  float3 result = Kd * ambient_light_color;

  // compute direct lighting
  unsigned int num_lights = lights.size();
  for(int i = 0; i < num_lights; ++i) {
    BasicLight light = lights[i];
    float Ldist = optix::length(light.pos - hit_point);
    float3 L = optix::normalize(light.pos - hit_point);
    float nDl = optix::dot( normal, L);

    // cast shadow ray
    float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
    if ( nDl > 0.0f && light.casts_shadow && show_shadows == 1) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      optix::Ray shadow_ray = optix::make_Ray(
          hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      light_attenuation = shadow_prd.attenuation;
    }

    // If not completely shadowed, light the hit point
    if( fmaxf(light_attenuation) > 0.0f ) {
      float3 Lc = light.color * light_attenuation;
      result += Kd * nDl * Lc;
    }
  }

  // pass the color back up the tree
  prd.result = result;
}
